/**
http://www.goldsborough.me/cuda/ml/cudnn/c++/2017/10/01/14-37-23-convolutions_with_cudnn/
this helped me get started with cudnn 
*/


#include <hipDNN.h>
#include "cudnn_conv.h"
#include "debug.h"
#include <iostream>


float cudnnConv(float* input, float* filter, float* output, int in_height, int in_width, int in_channels, int batch_size, int out_channels, int kernel_height, int kernel_width, int out_height, int out_width) {
  printf("here 0\n");
  hipdnnHandle_t cudnn;
  checkCUDNN(hipdnnCreate(&cudnn));

  hipdnnTensorDescriptor_t input_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NCHW, // can also be HIPDNN_TENSOR_NHWC
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/batch_size,
                                      /*channels=*/in_channels,
                                      /*image_height=*/in_height,
                                      /*image_width=*/in_width));

  hipdnnFilterDescriptor_t kernel_descriptor;
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*format=*/HIPDNN_TENSOR_NCHW,
                                      /*out_channels=*/out_channels,
                                      /*in_channels=*/in_channels,
                                      /*kernel_height=*/kernel_height,
                                      /*kernel_width=*/kernel_width));

  // TODO fix filter formatting, the kind we use isn't an option for cudnn
  hipdnnTensorDescriptor_t output_descriptor;
  checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                      /*format=*/HIPDNN_TENSOR_NCHW, // this means output_channels x input_channels x height x width
                                      /*dataType=*/HIPDNN_DATA_FLOAT,
                                      /*batch_size=*/batch_size,
                                      /*channels=*/out_channels,
                                      /*image_height=*/out_height,
                                      /*image_width=*/out_width));

  hipdnnConvolutionDescriptor_t convolution_descriptor;
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                           /*pad_height=*/0,
                                           /*pad_width=*/0,
                                           /*vertical_stride=*/1,
                                           /*horizontal_stride=*/1,
                                           /*dilation_height=*/1,
                                           /*dilation_width=*/1,
                                           /*mode=*/HIPDNN_CROSS_CORRELATION,
                                           /*computeType=*/HIPDNN_DATA_FLOAT));

  hipdnnConvolutionFwdAlgoPerf_t convolution_algorithm_perf;
  int returnedAlgoCount;
  checkCUDNN(cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
                                        input_descriptor,
                                        kernel_descriptor,
                                        convolution_descriptor,
                                        output_descriptor,
                                        1, // requested algo count
                                        &returnedAlgoCount,
                                        &convolution_algorithm_perf));
  hipdnnConvolutionFwdAlgo_t convolution_algorithm = convolution_algorithm_perf.algo;

  size_t workspace_bytes = 0;
  checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   convolution_descriptor,
                                                   output_descriptor,
                                                   convolution_algorithm,
                                                   &workspace_bytes));

  void* d_workspace{nullptr};
  checkCUDA(hipMalloc(&d_workspace, workspace_bytes));

  float elapsedTime; 
  hipEvent_t start, stop;
  checkCUDA( hipEventCreate( &start ) );
  checkCUDA( hipEventCreate( &stop ) );

  checkCUDA( hipEventRecord( start, 0 ) );

  const float alpha = 1, beta = 0;
  checkCUDNN(hipdnnConvolutionForward(cudnn,
                                   &alpha,
                                   input_descriptor,
                                   input,
                                   kernel_descriptor,
                                   filter,
                                   convolution_descriptor,
                                   convolution_algorithm,
                                   d_workspace,
                                   workspace_bytes,
                                   &beta,
                                   output_descriptor,
                                   output));
  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );
  return elapsedTime / 1000.0f;
}

